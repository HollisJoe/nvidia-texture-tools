#include "hip/hip_runtime.h"
// Copyright NVIDIA Corporation 2007 -- Ignacio Castano <icastano@nvidia.com>
// 
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use,
// copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following
// conditions:
// 
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
// OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
// WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
// OTHER DEALINGS IN THE SOFTWARE.

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "CudaMath.h"

#define NUM_THREADS 64		// Number of threads per block.

#if __DEVICE_EMULATION__
#define __debugsync() __syncthreads()
#else
#define __debugsync()
#endif

typedef unsigned short ushort;
typedef unsigned int uint;

template <class T> 
__device__ inline void swap(T & a, T & b)
{
	T tmp = a;
	a = b;
	b = tmp;
}

__constant__ float3 kColorMetric = { 1.0f, 1.0f, 1.0f };


////////////////////////////////////////////////////////////////////////////////
// Round color to RGB565 and expand
////////////////////////////////////////////////////////////////////////////////
inline __device__ float3 roundAndExpand(float3 v, ushort * w)
{
    v.x = rintf(__saturatef(v.x) * 31.0f);
    v.y = rintf(__saturatef(v.y) * 63.0f);
    v.z = rintf(__saturatef(v.z) * 31.0f);
    *w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
    v.x *= 0.03227752766457f; // approximate integer bit expansion.
    v.y *= 0.01583151765563f;
    v.z *= 0.03227752766457f;
    return v;
}


////////////////////////////////////////////////////////////////////////////////
// Evaluate permutations
////////////////////////////////////////////////////////////////////////////////
static __device__ float evalPermutation4(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = (1 + beta) / 3.0f;
        float alpha = 1.0f - beta;
    
        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
        betax_sum += beta * colors[i];
    }

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetric);
}


static __device__ float evalPermutation3(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = 0.5f;
        float alpha = 1.0f - beta;
    
        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
        betax_sum += beta * colors[i];
    }

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetric);
}

static __device__ float evalPermutation4(const float3 * colors, const float * weights, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);
		
        float beta = (bits & 1);
        if (bits & 2) beta = (1 + beta) / 3.0f;
        float alpha = 1.0f - beta;
		
		alpha2_sum += alpha * alpha * weights[i];
        beta2_sum += beta * beta * weights[i];
        alphabeta_sum += alpha * beta * weights[i];
        alphax_sum += alpha * colors[i] * weights[i];
        betax_sum += beta * colors[i] * weights[i];
    }

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetric);
}


static __device__ float evalPermutation3(const float3 * colors, const float * weights, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = 0.5f;
        float alpha = 1.0f - beta;
    
        alpha2_sum += alpha * alpha * weights[i];
        beta2_sum += beta * beta * weights[i];
        alphabeta_sum += alpha * beta * weights[i];
        alphax_sum += alpha * colors[i] * weights[i];
        betax_sum += beta * colors[i] * weights[i];
    }

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetric);
}


////////////////////////////////////////////////////////////////////////////////
// Sort colors
////////////////////////////////////////////////////////////////////////////////
__device__ void sortColors(float * values, int * cmp)
{
	int tid = threadIdx.x;

	cmp[tid] = (values[0] < values[tid]);
	cmp[tid] += (values[1] < values[tid]);
	cmp[tid] += (values[2] < values[tid]);
	cmp[tid] += (values[3] < values[tid]);
	cmp[tid] += (values[4] < values[tid]);
	cmp[tid] += (values[5] < values[tid]);
	cmp[tid] += (values[6] < values[tid]);
	cmp[tid] += (values[7] < values[tid]);
	cmp[tid] += (values[8] < values[tid]);
	cmp[tid] += (values[9] < values[tid]);
	cmp[tid] += (values[10] < values[tid]);
	cmp[tid] += (values[11] < values[tid]);
	cmp[tid] += (values[12] < values[tid]);
	cmp[tid] += (values[13] < values[tid]);
	cmp[tid] += (values[14] < values[tid]);
	cmp[tid] += (values[15] < values[tid]);
	
	// Resolve elements with the same index.
	if (tid > 0 && cmp[tid] == cmp[0]) ++cmp[tid];
	if (tid > 1 && cmp[tid] == cmp[1]) ++cmp[tid];
	if (tid > 2 && cmp[tid] == cmp[2]) ++cmp[tid];
	if (tid > 3 && cmp[tid] == cmp[3]) ++cmp[tid];
	if (tid > 4 && cmp[tid] == cmp[4]) ++cmp[tid];
	if (tid > 5 && cmp[tid] == cmp[5]) ++cmp[tid];
	if (tid > 6 && cmp[tid] == cmp[6]) ++cmp[tid];
	if (tid > 7 && cmp[tid] == cmp[7]) ++cmp[tid];
	if (tid > 8 && cmp[tid] == cmp[8]) ++cmp[tid];
	if (tid > 9 && cmp[tid] == cmp[9]) ++cmp[tid];
	if (tid > 10 && cmp[tid] == cmp[10]) ++cmp[tid];
	if (tid > 11 && cmp[tid] == cmp[11]) ++cmp[tid];
	if (tid > 12 && cmp[tid] == cmp[12]) ++cmp[tid];
	if (tid > 13 && cmp[tid] == cmp[13]) ++cmp[tid];
	if (tid > 14 && cmp[tid] == cmp[14]) ++cmp[tid];
}


////////////////////////////////////////////////////////////////////////////////
// Find index with minimum error
////////////////////////////////////////////////////////////////////////////////
__device__ void minimizeError(float * errors, int * indices)
{
	const int idx = threadIdx.x;

#if __DEVICE_EMULATION__
	for(int d = NUM_THREADS/2; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (idx < d)
		{
			float err0 = errors[idx];
			float err1 = errors[idx + d];
			
			if (err1 < err0) {
				errors[idx] = err1;
				indices[idx] = indices[idx + d];
			}
		}
	}

#else
	for(int d = NUM_THREADS/2; d > 32; d >>= 1)
	{
		__syncthreads();
		
		if (idx < d)
		{
			float err0 = errors[idx];
			float err1 = errors[idx + d];
			
			if (err1 < err0) {
				errors[idx] = err1;
				indices[idx] = indices[idx + d];
			}
		}
	}

	// unroll last 6 iterations
	if (idx <= 32)
	{
		if (errors[idx + 32] < errors[idx]) {
			errors[idx] = errors[idx + 32];
			indices[idx] = indices[idx + 32];
		}
		if (errors[idx + 16] < errors[idx]) {
			errors[idx] = errors[idx + 16];
			indices[idx] = indices[idx + 16];
		}
		if (errors[idx + 8] < errors[idx]) {
			errors[idx] = errors[idx + 8];
			indices[idx] = indices[idx + 8];
		}
		if (errors[idx + 4] < errors[idx]) {
			errors[idx] = errors[idx + 4];
			indices[idx] = indices[idx + 4];
		}
		if (errors[idx + 2] < errors[idx]) {
			errors[idx] = errors[idx + 2];
			indices[idx] = indices[idx + 2];
		}
		if (errors[idx + 1] < errors[idx]) {
			errors[idx] = errors[idx + 1];
			indices[idx] = indices[idx + 1];
		}
	}
#endif
}


////////////////////////////////////////////////////////////////////////////////
// Load color block to shared mem
////////////////////////////////////////////////////////////////////////////////
__device__ void loadColorBlock(const uint * image, float3 colors[16], int xrefs[16])
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	__shared__ float dps[16];

	if (idx < 16)
	{
		// Read color and copy to shared mem.
		uint c = image[(bid) * 16 + idx];
	
		colors[idx].z = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
		colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
		colors[idx].x = ((c >> 16) & 0xFF) * (1.0f / 255.0f);
		
		// No need to synchronize, 16 < warp size.
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif

		// Sort colors along the best fit line.
		float3 axis = bestFitLine(colors);
		
		dps[idx] = dot(colors[idx], axis);
		
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif
		
		sortColors(dps, xrefs);
		
		float3 tmp = colors[idx];
		colors[xrefs[idx]] = tmp;
	}
}

__device__ void loadColorBlock(const uint * image, float3 colors[16], float weights[16], int xrefs[16])
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	__shared__ float dps[16];

	if (idx < 16)
	{
		// Read color and copy to shared mem.
		uint c = image[(bid) * 16 + idx];
		
		colors[idx].z = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
		colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
		colors[idx].x = ((c >> 16) & 0xFF) * (1.0f / 255.0f);
		weights[idx] = ((c >> 24) & 0xFF) * (1.0f / 255.0f);
		
		// No need to synchronize, 16 < warp size.
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif

		// Sort colors along the best fit line.
		float3 axis = bestFitLine(colors);
		
		dps[idx] = dot(colors[idx], axis);
		
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif
		
		sortColors(dps, xrefs);
		
		float3 tmp = colors[idx];
		colors[xrefs[idx]] = tmp;
		
		float w = weights[idx];
		weights[xrefs[idx]] = tmp;
	}
}


__device__ void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16])
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	if (start == end)
	{
		permutation = 0;
	}
	
	// Reorder permutation.
	uint indices = 0;
	for(int i = 0; i < 16; i++)
	{
		int ref = xrefs[i];
		indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
	}
	
	// Write endpoints.
	result[bid].x = (end << 16) | start;
	
	// Write palette indices.
	result[bid].y = indices;
}


////////////////////////////////////////////////////////////////////////////////
// Compress color block
////////////////////////////////////////////////////////////////////////////////
__global__ void compress(const uint * permutations, const uint * image, uint2 * result)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;
	
	__shared__ float3 colors[16];
	__shared__ int xrefs[16];
	
	loadColorBlock(image, colors, xrefs);
	
	ushort bestStart, bestEnd;
	uint bestPermutation;
	float bestError = FLT_MAX;
	
	__syncthreads();

#if 0
	// This version is more clear, but slightly slower.
    for(int i = 0; i < 16; i++)
    {
        if (i == 15 && idx >= 32) break;
        
        ushort start, end;
        uint permutation = permutations[idx + NUM_THREADS * i];
        float error = evalPermutation4(colors, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;    // Flip indices.
    }

    for(int i = 0; i < 3; i++)
    {
        if (i == 2 && idx >= 32) break;
        
        ushort start, end;
        uint permutation = permutations[idx + NUM_THREADS * i];
        float error = evalPermutation3(colors, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
            
            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;    // Flip indices.
            }
        }
    }
#else
	{
		int pidx = idx + NUM_THREADS * 15;
		if (idx >= 32)
		{
			pidx = idx + NUM_THREADS * 2;
		}
		
		ushort start, end;
		uint permutation = permutations[pidx];
		float error = evalPermutation4(colors, permutation, &start, &end);
	    
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

    for(int i = 3; i < 15; i++)
    {
        ushort start, end;
        uint permutation = permutations[idx + NUM_THREADS * i];
        float error = evalPermutation4(colors, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;    // Flip indices.
	}


    for(int i = 0; i < 3; i++)
    {
        if (i == 2 && idx >= 32) break;
        
        ushort start, end;
        uint permutation = permutations[idx + NUM_THREADS * i];
        float error = evalPermutation3(colors, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
            
            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;    // Flip indices.
            }
        }
		
        error = evalPermutation4(colors, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
			
			if (bestStart < bestEnd)
			{
				swap(bestEnd, bestStart);
				bestPermutation ^= 0x55555555;    // Flip indices.
			}
        }
    }
#endif
	
	__syncthreads();
	
	// Use a parallel reduction to find minimum error.
	__shared__ float errors[NUM_THREADS];
	__shared__ int indices[NUM_THREADS];
	
	errors[idx] = bestError;
	indices[idx] = idx;
	
	minimizeError(errors, indices);
	
	__syncthreads();
	
	// Only write the result of the winner thread.
	if (idx == indices[0])
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs);
	}
}


__global__ void compressWeighted(const uint * permutations, const uint * image, uint2 * result)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;
	
	__shared__ float3 colors[16];
	__shared__ float weights[16];
	__shared__ int xrefs[16];
	
	loadColorBlock(image, colors, weights, xrefs);

	ushort bestStart, bestEnd;
	uint bestPermutation;
	float bestError = FLT_MAX;
	
	__syncthreads();

#if 0
	// This version is more clear, but slightly slower.
    for(int i = 0; i < 16; i++)
    {
        if (i == 15 && idx >= 32) break;
        
        ushort start, end;
        uint permutation = permutations[idx + NUM_THREADS * i];
        float error = evalPermutation4(colors, weights, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;    // Flip indices.
    }

    for(int i = 0; i < 3; i++)
    {
        if (i == 2 && idx >= 32) break;
        
        ushort start, end;
        uint permutation = permutations[idx + NUM_THREADS * i];
        float error = evalPermutation3(colors, weights, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
            
            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;    // Flip indices.
            }
        }
    }
#else
	{
		int pidx = idx + NUM_THREADS * 15;
		if (idx >= 32)
		{
			pidx = idx + NUM_THREADS * 2;
		}
		
		ushort start, end;
		uint permutation = permutations[pidx];
		float error = evalPermutation4(colors, weights, permutation, &start, &end);
	    
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

    for(int i = 3; i < 15; i++)
    {
        ushort start, end;
        uint permutation = permutations[idx + NUM_THREADS * i];
        float error = evalPermutation4(colors, weights, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;    // Flip indices.
	}


    for(int i = 0; i < 3; i++)
    {
        if (i == 2 && idx >= 32) break;
        
        ushort start, end;
        uint permutation = permutations[idx + NUM_THREADS * i];
        float error = evalPermutation3(colors, weights, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
            
            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;    // Flip indices.
            }
        }
		
        error = evalPermutation4(colors, weights, permutation, &start, &end);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
			
			if (bestStart < bestEnd)
			{
				swap(bestEnd, bestStart);
				bestPermutation ^= 0x55555555;    // Flip indices.
			}
        }
    }
#endif
	
	__syncthreads();
	
	// Use a parallel reduction to find minimum error.
	__shared__ float errors[NUM_THREADS];
	__shared__ int indices[NUM_THREADS];
	
	errors[idx] = bestError;
	indices[idx] = idx;
	
	minimizeError(errors, indices);
	
	__syncthreads();
	
	// Only write the result of the winner thread.
	if (idx == indices[0])
	{
		if (bestStart == bestEnd)
		{
			bestPermutation = 0;
		}
		
		// Reorder permutation.
		uint perm = 0;
		for(int i = 0; i < 16; i++)
		{
			int ref = xrefs[i];
			perm |= ((bestPermutation >> (2 * ref)) & 3) << (2 * i);
		}
		
		// Write endpoints. (bestStart, bestEnd)
		result[bid].x = (bestEnd << 16) | bestStart;
		
		// Write palette indices (permutation).
		result[bid].y = perm;
	}
}



////////////////////////////////////////////////////////////////////////////////
// Setup kernel
////////////////////////////////////////////////////////////////////////////////

extern "C" void setupCompressKernel(const float weights[3])
{
	// Set constants.
	hipMemcpyToSymbol(HIP_SYMBOL(kColorMetric), weights, sizeof(float) * 3, 0);
}


////////////////////////////////////////////////////////////////////////////////
// Launch kernel
////////////////////////////////////////////////////////////////////////////////

extern "C" void compressKernel(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compress<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

extern "C" void compressWeightedKernel(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compressWeighted<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}
