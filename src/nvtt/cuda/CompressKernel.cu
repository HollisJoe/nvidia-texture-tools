#include "hip/hip_runtime.h"
// Copyright NVIDIA Corporation 2007 -- Ignacio Castano <icastano@nvidia.com>
// 
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use,
// copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following
// conditions:
// 
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
// OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
// WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
// OTHER DEALINGS IN THE SOFTWARE.

#include <math.h>
#include <float.h> // FLT_MAX

#include "CudaMath.h"

#include "../SingleColorLookup.h"

#define NUM_THREADS 64		// Number of threads per block.

#if __DEVICE_EMULATION__
#define __debugsync() __syncthreads()
#else
#define __debugsync()
#endif

typedef unsigned char uchar;
typedef unsigned short ushort;
typedef unsigned int uint;

template <class T> 
__device__ inline void swap(T & a, T & b)
{
	T tmp = a;
	a = b;
	b = tmp;
}

__constant__ float3 kColorMetric = { 1.0f, 1.0f, 1.0f };
__constant__ float3 kColorMetricSqr = { 1.0f, 1.0f, 1.0f };

// Some kernels read the input through texture.
texture<uchar4, 2, hipReadModeNormalizedFloat> tex;


////////////////////////////////////////////////////////////////////////////////
// Sort colors
////////////////////////////////////////////////////////////////////////////////
__device__ void sortColors(const float * values, int * ranks)
{
#if __DEVICE_EMULATION__
    if (threadIdx.x == 0)
    {
        for (int tid = 0; tid < 16; tid++)
        {
            int rank = 0;
            for (int i = 0; i < 16; i++)
            {
                rank += (values[i] < values[tid]);
            }
            
            ranks[tid] = rank;
        }

        // Resolve elements with the same index.
        for (int i = 0; i < 15; i++)
        {
            for (int tid = 0; tid < 16; tid++)
            {
                if (tid > i && ranks[tid] == ranks[i]) ++ranks[tid];
            }
        }
    }
#else
    const int tid = threadIdx.x;

    int rank = 0;

    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        rank += (values[i] < values[tid]);
    }
    
    ranks[tid] = rank;

    // Resolve elements with the same index.
    #pragma unroll
    for (int i = 0; i < 15; i++)
    {
        if (tid > i && ranks[tid] == ranks[i]) ++ranks[tid];
    }
#endif
}


////////////////////////////////////////////////////////////////////////////////
// Load color block to shared mem
////////////////////////////////////////////////////////////////////////////////
__device__ void loadColorBlock(const uint * image, float3 colors[16], float3 sums[16], int xrefs[16], int * sameColor)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	__shared__ float dps[16];

	if (idx < 16)
	{
		// Read color and copy to shared mem.
		uint c = image[(bid) * 16 + idx];
		
		colors[idx].z = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
		colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
		colors[idx].x = ((c >> 16) & 0xFF) * (1.0f / 255.0f);
		
		// No need to synchronize, 16 < warp size.
		__debugsync();
		
		// Sort colors along the best fit line.
		colorSums(colors, sums);
		float3 axis = bestFitLine(colors, sums[0], kColorMetric);
		
		*sameColor = (axis == make_float3(0, 0, 0));
		
		dps[idx] = dot(colors[idx], axis);
		
		__debugsync();
		
		sortColors(dps, xrefs);
		
		float3 tmp = colors[idx];
		__debugsync();
		colors[xrefs[idx]] = tmp;
	}
#if __DEVICE_EMULATION__
	else
	{
		__debugsync();
		__debugsync();
		__debugsync();
	}
#endif
}

__device__ void loadColorBlockTex(uint bn, uint w, float3 colors[16], float3 sums[16], int xrefs[16], int * sameColor)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	__shared__ float dps[16];

	if (idx < 16)
	{
		float x = 4 * ((bn + bid) % w) + idx % 4;
		float y = 4 * ((bn + bid) / w) + idx / 4;

		// Read color and copy to shared mem.
		float4 c = tex2D(tex, x, y);

		colors[idx].x = c.z;
		colors[idx].y = c.y;
		colors[idx].z = c.x;

		// No need to synchronize, 16 < warp size.
		__debugsync();
		
		// Sort colors along the best fit line.
		colorSums(colors, sums);
		float3 axis = bestFitLine(colors, sums[0], kColorMetric);
		
		*sameColor = (axis == make_float3(0, 0, 0));
		
		dps[idx] = dot(colors[idx], axis);
		
		__debugsync();
		
		sortColors(dps, xrefs);
		
		float3 tmp = colors[idx];
		__debugsync();
		colors[xrefs[idx]] = tmp;
	}
#if __DEVICE_EMULATION__
	else
	{
		__debugsync();
		__debugsync();
		__debugsync();
	}
#endif

}


__device__ void loadColorBlock(const uint * image, float3 colors[16], float3 sums[16], float weights[16], int xrefs[16], int * sameColor)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	__shared__ float3 rawColors[16];
	__shared__ float dps[16];

	if (idx < 16)
	{
		// Read color and copy to shared mem.
		uint c = image[(bid) * 16 + idx];
		
		rawColors[idx].z = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
		rawColors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
		rawColors[idx].x = ((c >> 16) & 0xFF) * (1.0f / 255.0f);
		weights[idx] = (((c >> 24) & 0xFF) + 1) * (1.0f / 256.0f);
		
		colors[idx] = rawColors[idx] * weights[idx];

		// No need to synchronize, 16 < warp size.
		__debugsync();

		// Sort colors along the best fit line.
		colorSums(colors, sums);
		float3 axis = bestFitLine(colors, sums[0], kColorMetric);

		*sameColor = (axis == make_float3(0, 0, 0));

		dps[idx] = dot(rawColors[idx], axis);
		
		__debugsync();
		
		sortColors(dps, xrefs);
		
		float3 tmp = colors[idx];
		float w = weights[idx];
		__debugsync();
		colors[xrefs[idx]] = tmp;
		weights[xrefs[idx]] = w;
	}
#if __DEVICE_EMULATION__
	else
	{
		__debugsync();
		__debugsync();
		__debugsync();
	}
#endif
}

__device__ void loadColorBlock(const uint * image, float2 colors[16], float2 sums[16], int xrefs[16], int * sameColor)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	__shared__ float dps[16];

	if (idx < 16)
	{
		// Read color and copy to shared mem.
		uint c = image[(bid) * 16 + idx];
	
		colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
		colors[idx].x = ((c >> 16) & 0xFF) * (1.0f / 255.0f);
		
		// No need to synchronize, 16 < warp size.
		__debugsync();

		// Sort colors along the best fit line.
		colorSums(colors, sums);
		float2 axis = bestFitLine(colors, sums[0]);
		
		*sameColor = (axis == make_float2(0, 0));

		dps[idx] = dot(colors[idx], axis);
		
		__debugsync();
		
		sortColors(dps, xrefs);
		
		float2 tmp = colors[idx];
		__debugsync();
		colors[xrefs[idx]] = tmp;
	}
#if __DEVICE_EMULATION__
	else
	{
		__debugsync();
		__debugsync();
		__debugsync();
	}
#endif
}


////////////////////////////////////////////////////////////////////////////////
// Round color to RGB565 and expand
////////////////////////////////////////////////////////////////////////////////
inline __device__ float3 roundAndExpand565(float3 v, ushort * w)
{
#if 0
	uint x = __float2uint_rn(__saturatef(v.x) * 31.0f);
	uint y = __float2uint_rn(__saturatef(v.y) * 63.0f);
	uint z = __float2uint_rn(__saturatef(v.z) * 31.0f);
	*w = (x << 11) | (y << 5) | z;
	v.x = __uint2float_rn(x) * 1.0f / 31.0f;
	v.y = __uint2float_rn(y) * 1.0f / 63.0f;
	v.z = __uint2float_rn(z) * 1.0f / 31.0f;
#else
	v.x = rintf(__saturatef(v.x) * 31.0f);
	v.y = rintf(__saturatef(v.y) * 63.0f);
	v.z = rintf(__saturatef(v.z) * 31.0f);
	*w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
	v.x *= 1.0f / 31.0f;
	v.y *= 1.0f / 63.0f;
	v.z *= 1.0f / 31.0f;
#endif
	return v;
}

inline __device__ float2 roundAndExpand56(float2 v, ushort * w)
{
	v.x = rintf(__saturatef(v.x) * 31.0f);
	v.y = rintf(__saturatef(v.y) * 63.0f);
	*w = ((ushort)v.x << 11) | ((ushort)v.y << 5);
	v.x *= 1.0f / 31.0f;
	v.y *= 1.0f / 63.0f;
	return v;
}

inline __device__ float2 roundAndExpand88(float2 v, ushort * w)
{
	v.x = rintf(__saturatef(v.x) * 255.0f);
	v.y = rintf(__saturatef(v.y) * 255.0f);
	*w = ((ushort)v.x << 8) | ((ushort)v.y);
	v.x *= 1.0f / 255.0f;
	v.y *= 1.0f / 255.0f;
	return v;
}


////////////////////////////////////////////////////////////////////////////////
// Evaluate permutations
////////////////////////////////////////////////////////////////////////////////
__device__ float evalPermutation4(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float alpha2_sum = 0.0f;
	float beta2_sum = 0.0f;
	float alphabeta_sum = 0.0f;
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
	float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

	// Compute alpha & beta for this permutation.
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);
		
		float beta = (bits & 1);
		if (bits & 2) beta = (1 + beta) / 3.0f;
		float alpha = 1.0f - beta;
		
		alpha2_sum += alpha * alpha;
		beta2_sum += beta * beta;
		alphabeta_sum += alpha * beta;
		alphax_sum += alpha * colors[i];
		betax_sum += beta * colors[i];
	}

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return dot(e, kColorMetricSqr);
}

__device__ float evalPermutation3(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float alpha2_sum = 0.0f;
	float beta2_sum = 0.0f;
	float alphabeta_sum = 0.0f;
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
	float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

	// Compute alpha & beta for this permutation.
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);

		float beta = (bits & 1);
		if (bits & 2) beta = 0.5f;
		float alpha = 1.0f - beta;
	
		alpha2_sum += alpha * alpha;
		beta2_sum += beta * beta;
		alphabeta_sum += alpha * beta;
		alphax_sum += alpha * colors[i];
		betax_sum += beta * colors[i];
	}

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return dot(e, kColorMetricSqr);
}

__constant__ const float alphaTable4[4] = { 9.0f, 0.0f, 6.0f, 3.0f };
__constant__ const float alphaTable3[4] = { 4.0f, 0.0f, 2.0f, 2.0f };
__constant__ const uint prods4[4] = { 0x090000,0x000900,0x040102,0x010402 };
__constant__ const uint prods3[4] = { 0x040000,0x000400,0x040101,0x010401 };

__device__ float evalPermutation4(const float3 * colors, float3 color_sum, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
	uint akku = 0;

	// Compute alpha & beta for this permutation.
	#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);
		
		alphax_sum += alphaTable4[bits & 3] * colors[i];
		akku += prods4[bits & 3];
	}

	float alpha2_sum = float(akku >> 16);
	float beta2_sum = float((akku >> 8) & 0xff);
	float alphabeta_sum = float(akku & 0xff);
	float3 betax_sum = 9.0f * color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return (1.0f / 9.0f) * dot(e, kColorMetricSqr);
}

__device__ float evalPermutation3(const float3 * colors, float3 color_sum, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
	uint akku = 0;

	// Compute alpha & beta for this permutation.
	#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);

		alphax_sum += alphaTable3[bits & 3] * colors[i];
		akku += prods3[bits & 3];
	}

	float alpha2_sum = float(akku >> 16);
	float beta2_sum = float((akku >> 8) & 0xff);
	float alphabeta_sum = float(akku & 0xff);
	float3 betax_sum = 4.0f * color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return (1.0f / 4.0f) * dot(e, kColorMetricSqr);
}

__device__ float evalPermutation4(const float3 * colors, const float * weights, float3 color_sum, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float alpha2_sum = 0.0f;
	float beta2_sum = 0.0f;
	float alphabeta_sum = 0.0f;
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

	// Compute alpha & beta for this permutation.
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);
		
		float beta = (bits & 1);
		if (bits & 2) beta = (1 + beta) / 3.0f;
		float alpha = 1.0f - beta;
		
		alpha2_sum += alpha * alpha * weights[i];
		beta2_sum += beta * beta * weights[i];
		alphabeta_sum += alpha * beta * weights[i];
		alphax_sum += alpha * colors[i];
	}

	float3 betax_sum = color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return dot(e, kColorMetricSqr);
}

/*
__device__ float evalPermutation3(const float3 * colors, const float * weights, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float alpha2_sum = 0.0f;
	float beta2_sum = 0.0f;
	float alphabeta_sum = 0.0f;
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

	// Compute alpha & beta for this permutation.
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);

		float beta = (bits & 1);
		if (bits & 2) beta = 0.5f;
		float alpha = 1.0f - beta;

		alpha2_sum += alpha * alpha * weights[i];
		beta2_sum += beta * beta * weights[i];
		alphabeta_sum += alpha * beta * weights[i];
		alphax_sum += alpha * colors[i];
	}

	float3 betax_sum = color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return dot(e, kColorMetricSqr);
}
*/

__device__ float evalPermutation4(const float2 * colors, float2 color_sum, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float2 alphax_sum = make_float2(0.0f, 0.0f);
	uint akku = 0;

	// Compute alpha & beta for this permutation.
	#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);
		
		alphax_sum += alphaTable4[bits & 3] * colors[i];
		akku += prods4[bits & 3];
	}

	float alpha2_sum = float(akku >> 16);
	float beta2_sum = float((akku >> 8) & 0xff);
	float alphabeta_sum = float(akku & 0xff);
	float2 betax_sum = 9.0f * color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float2 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float2 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6 color and expand...
	a = roundAndExpand56(a, start);
	b = roundAndExpand56(b, end);

	// compute the error
	float2 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return (1.0f / 9.0f) * (e.x + e.y);
}

__device__ float evalPermutation3(const float2 * colors, float2 color_sum, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float2 alphax_sum = make_float2(0.0f, 0.0f);
	uint akku = 0;

	// Compute alpha & beta for this permutation.
	#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);

		alphax_sum += alphaTable3[bits & 3] * colors[i];
		akku += prods3[bits & 3];
	}

	float alpha2_sum = float(akku >> 16);
	float beta2_sum = float((akku >> 8) & 0xff);
	float alphabeta_sum = float(akku & 0xff);
	float2 betax_sum = 4.0f * color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float2 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float2 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6 color and expand...
	a = roundAndExpand56(a, start);
	b = roundAndExpand56(b, end);

	// compute the error
	float2 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return (1.0f / 4.0f) * (e.x + e.y);
}

__device__ float evalPermutationCTX(const float2 * colors, float2 color_sum, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float2 alphax_sum = make_float2(0.0f, 0.0f);
	uint akku = 0;

	// Compute alpha & beta for this permutation.
	#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);
		
		alphax_sum += alphaTable4[bits & 3] * colors[i];
		akku += prods4[bits & 3];
	}

	float alpha2_sum = float(akku >> 16);
	float beta2_sum = float((akku >> 8) & 0xff);
	float alphabeta_sum = float(akku & 0xff);
	float2 betax_sum = 9.0f * color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float2 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float2 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 8-8 color and expand...
	a = roundAndExpand88(a, start);
	b = roundAndExpand88(b, end);

	// compute the error
	float2 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return (1.0f / 9.0f) * (e.x + e.y);
}


////////////////////////////////////////////////////////////////////////////////
// Evaluate all permutations
////////////////////////////////////////////////////////////////////////////////
__device__ void evalAllPermutations(const float3 * colors, float3 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	__shared__ uint s_permutations[160];

	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];
		if (pidx < 160) s_permutations[pidx] = permutation;
				
		float error = evalPermutation4(colors, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	for(int i = 0; i < 3; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 160) break;
		
		ushort start, end;
		uint permutation = s_permutations[pidx];
		float error = evalPermutation3(colors, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
			
			if (bestStart > bestEnd)
			{
				swap(bestEnd, bestStart);
				bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;	// Flip indices.
			}
		}
	}

	errors[idx] = bestError;
}

/*
__device__ void evalAllPermutations(const float3 * colors, const float * weights, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	__shared__ uint s_permutations[160];
	
	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];
		if (pidx < 160) s_permutations[pidx] = permutation;

		float error = evalPermutation4(colors, weights, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	for(int i = 0; i < 3; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 160) break;
		
		ushort start, end;
		uint permutation = s_permutations[pidx];
		float error = evalPermutation3(colors, weights, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
			
			if (bestStart > bestEnd)
			{
				swap(bestEnd, bestStart);
				bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;	// Flip indices.
			}
		}
	}

	errors[idx] = bestError;
}
*/

__device__ void evalAllPermutations(const float2 * colors, float2 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	__shared__ uint s_permutations[160];

	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];
		if (pidx < 160) s_permutations[pidx] = permutation;
				
		float error = evalPermutation4(colors, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	for(int i = 0; i < 3; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 160) break;
		
		ushort start, end;
		uint permutation = s_permutations[pidx];
		float error = evalPermutation3(colors, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
			
			if (bestStart > bestEnd)
			{
				swap(bestEnd, bestStart);
				bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;	// Flip indices.
			}
		}
	}

	errors[idx] = bestError;
}

__device__ void evalLevel4Permutations(const float3 * colors, float3 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];

		float error = evalPermutation4(colors, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	errors[idx] = bestError;
}

__device__ void evalLevel4Permutations(const float3 * colors, const float * weights, float3 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];

		float error = evalPermutation4(colors, weights, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	errors[idx] = bestError;
}

__device__ void evalAllPermutationsCTX(const float2 * colors, float2 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 704) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];

		float error = evalPermutationCTX(colors, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	errors[idx] = bestError;
}


////////////////////////////////////////////////////////////////////////////////
// Find index with minimum error
////////////////////////////////////////////////////////////////////////////////
__device__ int findMinError(float * errors)
{
	const int idx = threadIdx.x;

	__shared__ int indices[NUM_THREADS];
	indices[idx] = idx;

#if __DEVICE_EMULATION__
	for(int d = NUM_THREADS/2; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (idx < d)
		{
			float err0 = errors[idx];
			float err1 = errors[idx + d];
			
			if (err1 < err0) {
				errors[idx] = err1;
				indices[idx] = indices[idx + d];
			}
		}
	}
#else
	for(int d = NUM_THREADS/2; d > 32; d >>= 1)
	{
		__syncthreads();
		
		if (idx < d)
		{
			float err0 = errors[idx];
			float err1 = errors[idx + d];
			
			if (err1 < err0) {
				errors[idx] = err1;
				indices[idx] = indices[idx + d];
			}
		}
	}

	__syncthreads();

	// unroll last 6 iterations
	if (idx < 32)
	{
		if (errors[idx + 32] < errors[idx]) {
			errors[idx] = errors[idx + 32];
			indices[idx] = indices[idx + 32];
		}
		if (errors[idx + 16] < errors[idx]) {
			errors[idx] = errors[idx + 16];
			indices[idx] = indices[idx + 16];
		}
		if (errors[idx + 8] < errors[idx]) {
			errors[idx] = errors[idx + 8];
			indices[idx] = indices[idx + 8];
		}
		if (errors[idx + 4] < errors[idx]) {
			errors[idx] = errors[idx + 4];
			indices[idx] = indices[idx + 4];
		}
		if (errors[idx + 2] < errors[idx]) {
			errors[idx] = errors[idx + 2];
			indices[idx] = indices[idx + 2];
		}
		if (errors[idx + 1] < errors[idx]) {
			errors[idx] = errors[idx + 1];
			indices[idx] = indices[idx + 1];
		}
	}
#endif

	__syncthreads();

	return indices[0];
}


////////////////////////////////////////////////////////////////////////////////
// Save DXT block
////////////////////////////////////////////////////////////////////////////////
__device__ void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 * result)
{
	const int bid = blockIdx.x;

	if (start == end)
	{
		permutation = 0;
	}
	
	// Reorder permutation.
	uint indices = 0;
	for(int i = 0; i < 16; i++)
	{
		int ref = xrefs[i];
		indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
	}
	
	// Write endpoints.
	result[bid].x = (end << 16) | start;
	
	// Write palette indices.
	result[bid].y = indices;
}

__device__ void saveBlockCTX1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 * result)
{
	saveBlockDXT1(start, end, permutation, xrefs, result);
}

__device__ void saveSingleColorBlockDXT1(float3 color, uint2 * result)
{
	const int bid = blockIdx.x;

	int r = color.x * 255;
	int g = color.y * 255;
	int b = color.z * 255;

	ushort color0 = (OMatch5[r][0] << 11) | (OMatch6[g][0] << 5) | OMatch5[b][0];
	ushort color1 = (OMatch5[r][1] << 11) | (OMatch6[g][1] << 5) | OMatch5[b][1];

	if (color0 < color1)
	{
		result[bid].x = (color0 << 16) | color1;
		result[bid].y = 0xffffffff;
	}
	else
	{
		result[bid].x = (color1 << 16) | color0;
		result[bid].y = 0xaaaaaaaa;
	}
}

__device__ void saveSingleColorBlockDXT1(float2 color, uint2 * result)
{
	const int bid = blockIdx.x;

	int r = color.x * 255;
	int g = color.y * 255;

	ushort color0 = (OMatch5[r][0] << 11) | (OMatch6[g][0] << 5);
	ushort color1 = (OMatch5[r][1] << 11) | (OMatch6[g][1] << 5);

	if (color0 < color1)
	{
		result[bid].x = (color0 << 16) | color1;
		result[bid].y = 0xffffffff;
	}
	else
	{
		result[bid].x = (color1 << 16) | color0;
		result[bid].y = 0xaaaaaaaa;
	}
}

__device__ void saveSingleColorBlockCTX1(float2 color, uint2 * result)
{
	const int bid = blockIdx.x;

	int r = color.x * 255;
	int g = color.y * 255;

	ushort color0 = (r << 8) | (g);
	
	result[bid].x = (color0 << 16) | color0;
	result[bid].y = 0x00000000;
}


////////////////////////////////////////////////////////////////////////////////
// Compress color block
////////////////////////////////////////////////////////////////////////////////
__global__ void compressDXT1(const uint * permutations, const uint * image, uint2 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ int xrefs[16];
	__shared__ int sameColor;
	
	loadColorBlock(image, colors, sums, xrefs, &sameColor);

	__syncthreads();

	if (sameColor)
	{
		if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
		return;
	}

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];

	evalAllPermutations(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	const int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
	}
}

__global__ void compressDXT1_Tex(uint bn, uint w, const uint * permutations, uint2 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ int xrefs[16];
	__shared__ int sameColor;
	
	loadColorBlockTex(bn, w, colors, sums, xrefs, &sameColor);

	__syncthreads();

	if (sameColor)
	{
		if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
		return;
	}

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];

	evalAllPermutations(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	const int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
	}
}


__global__ void compressLevel4DXT1(const uint * permutations, const uint * image, uint2 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ int xrefs[16];
	__shared__ int sameColor;
	
	loadColorBlock(image, colors, sums, xrefs, &sameColor);

	__syncthreads();

	if (sameColor)
	{
		if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
		return;
	}

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];

	evalLevel4Permutations(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	const int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
	}
}

__global__ void compressWeightedDXT1(const uint * permutations, const uint * image, uint2 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ float weights[16];
	__shared__ int xrefs[16];
	__shared__ int sameColor;
	
	loadColorBlock(image, colors, sums, weights, xrefs, &sameColor);
	
	__syncthreads();

	if (sameColor)
	{
		if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
		return;
	}

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];
	
	evalLevel4Permutations(colors, weights, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
	}
}


__global__ void compressNormalDXT1(const uint * permutations, const uint * image, uint2 * result)
{
	__shared__ float2 colors[16];
	__shared__ float2 sums[16];
	__shared__ int xrefs[16];
	__shared__ int sameColor;

	loadColorBlock(image, colors, sums, xrefs, &sameColor);
	
	__syncthreads();

	if (sameColor)
	{
		if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
		return;
	}

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];

	evalAllPermutations(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	const int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
	}
}

__global__ void compressCTX1(const uint * permutations, const uint * image, uint2 * result)
{
	__shared__ float2 colors[16];
	__shared__ float2 sums[16];
	__shared__ int xrefs[16];
	__shared__ int sameColor;
	
	loadColorBlock(image, colors, sums, xrefs, &sameColor);
	
	__syncthreads();

	if (sameColor)
	{
		if (threadIdx.x == 0) saveSingleColorBlockCTX1(colors[0], result);
		return;
	}

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];

	evalAllPermutationsCTX(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	const int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockCTX1(bestStart, bestEnd, bestPermutation, xrefs, result);
	}
}


/*
__device__ float computeError(const float weights[16], uchar a0, uchar a1)
{
	float palette[6];
	palette[0] = (6.0f/7.0f * a0 + 1.0f/7.0f * a1);
	palette[1] = (5.0f/7.0f * a0 + 2.0f/7.0f * a1);
	palette[2] = (4.0f/7.0f * a0 + 3.0f/7.0f * a1);
	palette[3] = (3.0f/7.0f * a0 + 4.0f/7.0f * a1);
	palette[4] = (2.0f/7.0f * a0 + 5.0f/7.0f * a1);
	palette[5] = (1.0f/7.0f * a0 + 6.0f/7.0f * a1);

	float total = 0.0f;

	for (uint i = 0; i < 16; i++)
	{
		float alpha = weights[i];

		float error = a0 - alpha;
		error = min(error, palette[0] - alpha);
		error = min(error, palette[1] - alpha);
		error = min(error, palette[2] - alpha);
		error = min(error, palette[3] - alpha);
		error = min(error, palette[4] - alpha);
		error = min(error, palette[5] - alpha);
		error = min(error, a1 - alpha);
		
		total += error;
	}
	
	return total;
}

inline __device__ uchar roundAndExpand(float a)
{
	return rintf(__saturatef(a) * 255.0f);
}
*/
/*
__device__ void optimizeAlpha8(const float alphas[16], uchar & a0, uchar & a1)
{
	float alpha2_sum = 0;
	float beta2_sum = 0;
	float alphabeta_sum = 0;
	float alphax_sum = 0;
	float betax_sum = 0;

	for (int i = 0; i < 16; i++)
	{
		uint idx = index[i];
		float alpha;
		if (idx < 2) alpha = 1.0f - idx;
		else alpha = (8.0f - idx) / 7.0f;
		
		float beta = 1 - alpha;

		alpha2_sum += alpha * alpha;
		beta2_sum += beta * beta;
		alphabeta_sum += alpha * beta;
		alphax_sum += alpha * alphas[i];
		betax_sum += beta * alphas[i];
	}

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

	a0 = roundAndExpand8(a);
	a1 = roundAndExpand8(b);
}
*/
/*
__device__ void compressAlpha(const float alphas[16], uint4 * result)
{
	const int tid = threadIdx.x;
	
	// Compress alpha block!
	// Brute force approach:
	// Try all color pairs: 256*256/2 = 32768, 32768/64 = 512 iterations?

	// Determine min & max alphas

	float A0, A1;

	if (tid < 16)
	{
		__shared__ uint s_alphas[16];
		
		s_alphas[tid] = alphas[tid];
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^8]);
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^4]);
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^2]);
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^1]);
		A0 = s_alphas[tid];
		
		s_alphas[tid] = alphas[tid];
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^8]);
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^4]);
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^2]);
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^1]);
		A1 = s_alphas[tid];
	}

	__syncthreads();

	int minIdx = 0;

	if (A1 - A0 > 8)
	{
		float bestError = FLT_MAX;

		// 64 threads -> 8x8
		// divide [A1-A0] in partitions.
		// test endpoints 
		
		for (int i = 0; i < 128; i++)
		{
			uint idx = (i * NUM_THREADS + tid) * 4;
			uchar a0 = idx & 255;
			uchar a1 = idx >> 8;
			
			float error = computeError(alphas, a0, a1);
			
			if (error < bestError)
			{
				bestError = error;
				A0 = a0;
				A1 = a1;
			}
		}
		
		__shared__ float errors[NUM_THREADS];
		errors[tid] = bestError;
		
		// Minimize error.
		minIdx = findMinError(errors);

	}

	if (minIdx == tid)
	{
		// @@ Compute indices.
	
		// @@ Write alpha block.
	}
}

__global__ void compressDXT5(const uint * permutations, const uint * image, uint4 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ float weights[16];
	__shared__ int xrefs[16];
	
	loadColorBlock(image, colors, sums, weights, xrefs);
	
	__syncthreads();

	compressAlpha(weights, result);	

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];
	
	evalLevel4Permutations(colors, weights, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, (uint2 *)result);
	}
}
*/

/*__device__ void evaluatePalette(uint alpha0, uint alpha1, uint alphas[8])
{
	alpha[0] = alpha0;
	alpha[1] = alpha1;
	alpha[2] = (6 * alpha[0] + 1 * alpha[1]) / 7;	// bit code 010
	alpha[3] = (5 * alpha[0] + 2 * alpha[1]) / 7;	// bit code 011
	alpha[4] = (4 * alpha[0] + 3 * alpha[1]) / 7;	// bit code 100
	alpha[5] = (3 * alpha[0] + 4 * alpha[1]) / 7;	// bit code 101
	alpha[6] = (2 * alpha[0] + 5 * alpha[1]) / 7;	// bit code 110
	alpha[7] = (1 * alpha[0] + 6 * alpha[1]) / 7;	// bit code 111
}

__device__ uint computeAlphaError(const uint block[16], uint alpha0, uint alpha1, int bestError = INT_MAX)
{
	uint8 alphas[8];
	evaluatePalette(alpha0, alpha1, alphas);

	int totalError = 0;

	for (uint i = 0; i < 16; i++)
	{
		uint8 alpha = block[i];

		// @@ It should be possible to do this much faster.

		int minDist = INT_MAX;
		for (uint p = 0; p < 8; p++)
		{
			int dist = alphaDistance(alpha, alphas[p]);
			minDist = min(dist, minDist);
		}



		totalError += minDist;

		if (totalError > bestError)
		{
			// early out
			return totalError;
		}
	}

	return totalError;
}


void compressDXT5A(uint alpha[16])
{
	// Get min/max alpha.
	for (uint i = 0; i < 16; i++)
	{
		mina = min(mina, alpha[i]);
		maxa = max(maxa, alpha[i]);
	}

	dxtBlock->alpha0 = maxa;
	dxtBlock->alpha1 = mina;

	if (maxa - mina > 8)
	{
		int besterror = computeAlphaError(rgba, dxtBlock);
		int besta0 = maxa;
		int besta1 = mina;

		// Expand search space a bit.
		const int alphaExpand = 8;
		mina = (mina <= alphaExpand) ? 0 : mina - alphaExpand;
		maxa = (maxa <= 255-alphaExpand) ? 255 : maxa + alphaExpand;

		for (int a0 = mina+9; a0 < maxa; a0++)
		{
			for (int a1 = mina; a1 < a0-8; a1++)
			{
				nvDebugCheck(a0 - a1 > 8);

				dxtBlock->alpha0 = a0;
				dxtBlock->alpha1 = a1;
				int error = computeAlphaError(rgba, dxtBlock, besterror);

				if (error < besterror)
				{
					besterror = error;
					besta0 = a0;
					besta1 = a1;
				}
			}
		}

		dxtBlock->alpha0 = besta0;
		dxtBlock->alpha1 = besta1;
	}
}

__global__ void compressDXT5n(uint blockNum, uint2 * d_result)
{
	uint idx = blockIdx.x * 128 + threadIdx.x;

	if (idx >= blockNum)
	{
		return;
	}

	// @@ Ideally we would load the data to shared mem to achieve coalesced global mem access.
	// @@ Blocks would require too much shared memory (8k) and limit occupancy.

	// @@ Ideally we should use SIMD processing, multiple threads (4-8) processing the same block.
	// That simplifies coalescing, and reduces divergence.

	// @@ Experiment with texture. That's probably the most simple approach.

	uint x[16];
	uint y[16];


}
*/


////////////////////////////////////////////////////////////////////////////////
// Setup kernel
////////////////////////////////////////////////////////////////////////////////

extern "C" void setupCompressKernel(const float weights[3])
{
	// Set constants.
	hipMemcpyToSymbol(HIP_SYMBOL(kColorMetric), weights, sizeof(float) * 3, 0);

	float weightsSqr[3];
	weightsSqr[0] = weights[0] * weights[0];
	weightsSqr[1] = weights[1] * weights[1];
	weightsSqr[2] = weights[2] * weights[2];

	hipMemcpyToSymbol(HIP_SYMBOL(kColorMetricSqr), weightsSqr, sizeof(float) * 3, 0);
}


////////////////////////////////////////////////////////////////////////////////
// Launch kernel
////////////////////////////////////////////////////////////////////////////////

extern "C" void compressKernelDXT1(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compressDXT1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

extern "C" void compressKernelDXT1_Tex(uint bn, uint blockNum, uint w, hipArray * d_data, uint * d_result, uint * d_bitmaps)
{
	// Setup texture
	tex.normalized = false;
	tex.filterMode = hipFilterModePoint;
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
    
	hipBindTextureToArray(tex, d_data);

	compressDXT1_Tex<<<blockNum, NUM_THREADS>>>(bn, w, d_bitmaps, (uint2 *)d_result);
}


extern "C" void compressKernelDXT1_Level4(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compressLevel4DXT1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

extern "C" void compressWeightedKernelDXT1(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compressWeightedDXT1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

extern "C" void compressNormalKernelDXT1(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compressNormalDXT1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

extern "C" void compressKernelCTX1(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compressCTX1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

extern "C" void compressKernelDXT5n(uint blockNum, hipArray * d_data, uint * d_result)
{
	// Setup texture
	tex.normalized = false;
	tex.filterMode = hipFilterModePoint;
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
    
	hipBindTextureToArray(tex, d_data);

//	compressDXT5n<<<blockNum/128, 128>>>(blockNum, (uint2 *)d_result);
}
